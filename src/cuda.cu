#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <limits.h>
#include <fstream>

// Number of vertices
int N = 0;

void printGraph(int *arr, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%d ", arr[i * size + j]);
        }
        printf("\n");
    }
}

__global__ void initializeVisited(int *result, bool *visited) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    visited[index] = false;
    
    if ( index == ((blockDim.x * blockIdx.x) + blockIdx.x)) {
        result[index] = 0;
    } else {
        else result[index] = INT_MAX;
    }
}

__global__ void dijkstra(int *graph, int *result, bool* visited, int N) {
    
   for (int i = 0; i < N-1; i++) {
       // Get vertex with minimum distance
       int blockIndex1D = N  * blockIdx.x;
       int minDistance = INT_MAX;
       int minVertex;

       for (int vertex = 0; vertex < N; vertex++) {
           if (!visited[blockIndex1D + vertex] && result[blockIndex1D +  vertex] <= minDistance) {
                minDistance = result[blockIndex1D + vertex];
                minVertex = vertex;
           }
       }

       visited[blockIndex1D + minVertex] = true;
       int minBlockIndex1D = N * minVertex;
       
       for (int vertex = 0; vertex < N; vertex++) {
           if (!visited[blockIndex1D + vertex] &&
                graph[minBlockIndex1D + vertex] &&
                result[blockIndex1D + minVertex] != INT_MAX &&
                result[blockIndex1D + minVertex] + graph[minBlockIndex1D + vertex] < result[blockIndex1D + vertex]) {
                    result[blockIndex1D + vertex] = result[blockIndex1D + minVertex] + graph[minBlockIndex1D + vertex];
                }
       }
   }
}

int main(int argc, char *argv[]) {

    // Get matrix size from argument vector in , convert to int
    N = strtol(argv[1], NULL, 10);
    printf("N: %d\n ", N);

    int* cpuGraph = (int *) malloc(sizeof(int) * N * N);
    int* result = (int *) malloc(N * N * sizeof(int));
    
    srand(13517115);
    // Fill the matrix with rand() function
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cpuGraph[i * N + j] = rand() % 1000;
            if (i == j) {
                cpuGraph[i * N + j] = 0;
            }
        }
    }

    // Variable declaration for measuring time
    float totalTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *gpuGraph, *gpuResult;
    bool *gpuVisited;

    hipMalloc((void **) &gpuGraph, (sizeof(int) * N * N));
    hipMalloc((void **) &gpuVisited, (sizeof(bool) * N * N));
    hipMalloc((void **) &gpuResult, (sizeof(int) * N * N));
    
    hipEventRecord(start);
    // Copy from cpuGraph to gpuGraph (transfer from cpu to gpu!)
    hipMemcpy(gpuGraph, cpuGraph, (sizeof(int) * N * N), hipMemcpyHostToDevice);

    // Initialize visited graph: dimGrid = N, dimBlock = N
    initializeVisited<<<N, N>>>(gpuResult, gpuVisited);
    // Do the dijkstra: dimGrid = N, dimBlock = 1 (only 1 thread per block)
    dijkstra<<<N, 1>>>(gpuGraph, gpuResult, gpuVisited, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&totalTime, start, stop);

    // Copy from gpuGraph to cpuGraph (transfer from gpu to cpu!)
    hipMemcpy(result, gpuResult, (sizeof(int) * N * N), hipMemcpyDeviceToHost);
    // Print elapsed time in microsecs
    printf("%f µs\n", totalTime * 1000);

    char filename[100];
    snprintf(filename, sizeof(char) * 100, "output-%i.txt", N);
    FILE *f = fopen(filename, "w");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(f, "%d ", result[i * N + j]);
        }
        fprintf(f, "\n");
    }
    fclose(f);

    hipFree(gpuResult);
    hipFree(gpuVisited);
    hipFree(gpuGraph);
    free(cpuGraph);
    free(result);

    return 0;
}
